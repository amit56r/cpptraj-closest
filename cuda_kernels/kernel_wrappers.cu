#include "hip/hip_runtime.h"

#include <cstdio>



// device kernel def
__global__ void Action_noImage_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms);

////////////////////////





void Action_NoImage_Center(double *SolventMols_,double *D_, double maskCenter[3],double maxD,int  NMols, int NAtoms, float &time_gpu)
{


  hipEvent_t start_event, stop_event;
  float elapsed_time_gpu;

  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  int t4;
  int t2;
  double Dist;
  int solventMol;
  int solventAtom;



  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));
  hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);
  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);


  dim3 dimGrid0 = dim3(NMols,1);
  dim3 dimBlock0 = dim3(NAtoms,1);


  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms); 
  printf("About to launch kernel.\n");


  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);

  Action_noImage_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD, NMols, NAtoms);
  
  hipDeviceSynchronize();
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event );


  printf("Done with kernel CUDA Kernel Time: %.2f\n", elapsed_time_gpu);

  time_gpu  = elapsed_time_gpu;
  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
}
