#include "hip/hip_runtime.h"

#include <cstdio>
#include <cmath>

#define BLOCKDIM 1024


// device kernel def
__global__ void Action_noImage_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms, int active_size);
__global__ void Action_noImage_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, int Nmols , int NAtoms,int NSAtoms , int active_size);


//for imaging with ortho
__global__ void Action_ImageOrtho_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, double *box, int Nmols , int NAtoms, int active_size);
__global__ void Action_ImageOrtho_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, double *box, int Nmols , int NAtoms,int NSAtoms , int active_size);

//for imaging with NONortho
//TODO

////////////////////////





void Action_NoImage_Center(double *SolventMols_,double *D_, double maskCenter[3],double maxD,int  NMols, int NAtoms, float &time_gpu,int type, double box[3])
{


  hipEvent_t start_event, stop_event;
  float elapsed_time_gpu;

  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  double *boxDev;
  int t4;
  int t2;
  double Dist;
  int solventMol;
  int solventAtom;



  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));
  
  hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);
  
  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
  hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);



  //figue out the decomposition here
  //we need to pad as well

 // due to lack to  using center, each thread is going  rocess the solvent mol 
 //instead of atoms (make it alot easier)   (speacially for the imaging case)




  //figure out how many active thread in a block
  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  //int NBlocks =  ceil(NMols * NAtoms / float(active_size));  //having unroll factor
  int NBlocks = ceil(float(NMols)/ BLOCKDIM);

  // printf("Nmols = %d; Natoms = %d\n", NMols, NAtoms);
  // printf("active_size =  %d\n", active_size);
  //printf("NBlocks =  %d\n", NBlocks);
  //printf("sezeof(double) = %d\n", sizeof(double));
  //exit(0);



  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);


  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms); 
  printf("About to launch kernel.\n");


  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);

  if(type == 0)
    Action_noImage_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD, NMols, NAtoms,active_size);
  else if (type == 1)
    Action_ImageOrtho_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD,boxDev, NMols, NAtoms,active_size);
  else
    printf("kernel_wrapper: error in type\n");

  
  hipDeviceSynchronize();
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event );


  printf("Done with kernel CUDA Kernel Time: %.2f\n", elapsed_time_gpu);

  time_gpu  = elapsed_time_gpu;
  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
  hipFree(boxDev);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Action_NoImage_no_Center(double *SolventMols_,double *D_, double *Solute_atoms,double maxD,int  NMols, int NAtoms,int NSAtoms, float &time_gpu, int type,double box[3])
{


  hipEvent_t start_event, stop_event;
  float elapsed_time_gpu;

  double *devI3Ptr;
  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  double *boxDev;
  int t4;
  int t2;
  double Dist;
  int solventMol;
  int solventAtom;



  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));

  //hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  //hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);
  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);
  
  hipMalloc(((void **)(&devI3Ptr)), NSAtoms * 3 * sizeof(double ));
  hipMemcpy(devI3Ptr,Solute_atoms,NSAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
  hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);



  //figue out the decomposition here
  //we need to pad as well

  //figure out how many active thread in a block
  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  int NBlocks =  ceil(NMols * NAtoms / float(active_size));
  // printf("Nmols = %d; Natoms = %d\n", NMols, NAtoms);
  // printf("active_size =  %d\n", active_size);
  // printf("NBlocks =  %d\n", NBlocks);
  //printf("sezeof(double) = %d\n", sizeof(double));
  //exit(0);



  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);


  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms); 
  printf("About to launch kernel.\n");


  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);

  if(type == 0)
    Action_noImage_no_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, NMols, NAtoms,NSAtoms,active_size);
  else if(type == 1)
    Action_ImageOrtho_no_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, boxDev,  NMols, NAtoms,NSAtoms,active_size);
  else
    printf("kernel_wrapper: error in type no center version\n");
  
  hipDeviceSynchronize();
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event );


  printf("Done with kernel CUDA Kernel Time: %.2f\n", elapsed_time_gpu);

  time_gpu  = elapsed_time_gpu;
  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  //hipFree(devI1Ptr);
  hipFree(devI2Ptr);
  hipFree(devI3Ptr);
  hipFree(boxDev);
}