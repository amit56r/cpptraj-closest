
#include <hip/hip_runtime.h>
#include <stdio.h>




__global__ void Action_noImage_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms)
{
  int bx;
  bx = blockIdx.x;
  int tx;
  tx = threadIdx.x;
  double Dist;
  int t2;
  int t4;

  if(tx == 0 && bx == 0)
  	D_[bx] = maxD;
  __syncthreads();


  int sIndex =  bx*NAtoms*3 + tx*3;

  double x =  maskCenter[0] - SolventMols_[sIndex + 0];
  double y = maskCenter[1] - SolventMols_[sIndex + 1];
  double z =  maskCenter[2] - SolventMols_[sIndex + 2];
  Dist = x*x + y*y + z*z;
  if (Dist  < D_[bx]) 
    D_[bx] = Dist;

  if(tx == 0 && bx == 0 )
	printf("end of kernel");
}
