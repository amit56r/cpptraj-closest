
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM 1024
#define RSIZE 1024
#define C_FACTOR 4

//------------------------------------------------------------------------------------------------------------------------------------------------
//try thread coarsening 

__global__ void Action_noImage_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms, int active_size)
{

	__shared__ double dist_array[BLOCKDIM];



	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	//advantage of register
	double a0 = maskCenter[0];
	double a1 = maskCenter[1];
	double a2 = maskCenter[2];



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads();


		int sIndex =  mol*NAtoms*3 + atom*3;

		double x =  a0 - SolventMols_[sIndex + 0];
		double y = a1 - SolventMols_[sIndex + 1];
		double z =  a2 - SolventMols_[sIndex + 2];
	//Dist = x*x + y*y + z*z;
		dist_array[threadIdx.x] = x*x + y*y + z*z;
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		int i;



		double min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}


	// int i;
	// 	double min_val  = maxD;
	// 	if( atom ==0 )
	// 	{
	// 		for(i  = 0 ; i < NAtoms ; i++ ){
	// 			//sIndex = mol*NAtoms*3 + i*3;
	// 			if (dist_array[threadIdx.x + i]  < min_val) 
	// 				min_val = dist_array[threadIdx.x + i] ;
	// 		}
	// 		D_[mol] = min_val;
	// 	}


	// double min_val  = maxD;
	// if( threadIdx.x < active_size/NAtoms )
	// {

	// 	for(i  = threadIdx.x*NAtoms ; i <threadIdx.x*NAtoms + NAtoms ; i++ ){
	// 		//sIndex = mol*NAtoms*3 + i*3;
	// 		if (dist_array[i]  < min_val) 
	// 			min_val = dist_array[i] ;
	// 	}
	// 	D_[blockIdx.x * active_size/NAtoms + threadIdx.x  ] = min_val;
	// }

//------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void Action_noImage_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, int Nmols , int NAtoms,int NSAtoms , int active_size)
{

	__shared__ double dist_array[BLOCKDIM];
	__shared__ double sAtom_shared[RSIZE];




	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	

	//handling the chunks for  solute_atoms
	int chunksize,start,end, NChunks,i,j;

	if(NSAtoms*3 > RSIZE)
	{
		chunksize = (RSIZE/3)*3;
		NChunks = ceil(double(NSAtoms*3)/chunksize);
		start = 0;
		end = chunksize;
	}
	else
	{
		chunksize = NSAtoms*3;
		NChunks = 1;
		start = 0;
		end = NSAtoms*3;
	}

	// if(threadIdx.x == 0 && blockIdx.x == 0 )
	// 	printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
	// 		chunksize,NChunks,start,end);



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads(); 
		double min_val  = maxD;
		double dist;
		int sIndex =  mol*NAtoms*3 + atom*3;
		double a0 = SolventMols_[sIndex + 0];
		double a1 = SolventMols_[sIndex + 1];
		double a2 = SolventMols_[sIndex + 2];


		for(i  = 0 ; i  < NChunks ; i++)
		{
			//copying to shared
			//if (threadIdx.x < (end - start))
			//	sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

			//__syncthreads();

			//TODO - add skew per thread 
			for (j = start ; j < end; j+=3 )
			{
				//int offset = start + (j + threadIdx.x)%(end - start);
				double x = Solute_atoms[j + 0]  - a0;
				double y = Solute_atoms[j + 1]  - a1;
				double z = Solute_atoms[j + 2]  - a2;
				dist =  x*x + y*y + z*z;
				//if (mol ==  11)
				//	printf("min  = %f\n",min_val);
				min_val = min(min_val,dist);


			}

			start = end;
			end = min(end + chunksize, NSAtoms*3);


		}

		dist_array[threadIdx.x] = min_val;
		//if (threadIdx.x == 0)
		//	printf("min_val  = %f\n",min_val);
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		//min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}





//------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Action_ImageOrtho_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, double *box, int Nmols , int NAtoms, int active_size)
{
	__shared__ double dist_array[BLOCKDIM];



	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	//advantage of register
	double a0 = maskCenter[0];
	double a1 = maskCenter[1];
	double a2 = maskCenter[2];



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads();


		int sIndex =  mol*NAtoms*3 + atom*3;

		double x =  a0 - SolventMols_[sIndex + 0];
		double y =  a1 - SolventMols_[sIndex + 1];
		double z =  a2 - SolventMols_[sIndex + 2];

		// Get rid of sign info
		if (x<0) x=-x;
		if (y<0) y=-y;
		if (z<0) z=-z;
		  // Get rid of multiples of box lengths 
		//TODO  WIERD that should be a way to simplify it
		while (x > box[0]) x = x - box[0];
		while (y > box[1]) y = y - box[1];
		while (z > box[2]) z = z - box[2];
		  // Find shortest distance in periodic reference
		double D = box[0] - x;
		if (D < x) x = D;
		D = box[1] - y;
		if (D < y) y = D;  
		D = box[2] - z;
		if (D < z) z = D;




	//Dist = x*x + y*y + z*z;
		dist_array[threadIdx.x] = x*x + y*y + z*z;
		if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
			dist_array[threadIdx.x] = -1.0;
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		int i;



		double min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}

//------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Action_ImageOrtho_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, double *box, int Nmols , int NAtoms,int NSAtoms , int active_size)
{

	__shared__ double dist_array[BLOCKDIM];
	__shared__ double sAtom_shared[RSIZE];




	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	

	//handling the chunks for  solute_atoms
	int chunksize,start,end, NChunks,i,j;

	if(NSAtoms*3 > RSIZE)
	{
		chunksize = (RSIZE/3)*3;
		NChunks = ceil(double(NSAtoms*3)/chunksize);
		start = 0;
		end = chunksize;
	}
	else
	{
		chunksize = NSAtoms*3;
		NChunks = 1;
		start = 0;
		end = NSAtoms*3;
	}

	// if(threadIdx.x == 0 && blockIdx.x == 0 )
	// 	printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
	// 		chunksize,NChunks,start,end);



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads(); 
		double min_val  = maxD;
		double dist;
		int sIndex =  mol*NAtoms*3 + atom*3;
		double a0 = SolventMols_[sIndex + 0];
		double a1 = SolventMols_[sIndex + 1];
		double a2 = SolventMols_[sIndex + 2];


		for(i  = 0 ; i  < NChunks ; i++)
		{
			//copying to shared
			//if (threadIdx.x < (end - start))
			//	sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

			//__syncthreads();

			//TODO - add skew per thread 
			for (j = start ; j < end; j+=3 )
			{
				//int offset = start + (j + threadIdx.x)%(end - start);
				double x = Solute_atoms[j + 0]  - a0;
				double y = Solute_atoms[j + 1]  - a1;
				double z = Solute_atoms[j + 2]  - a2;


				// Get rid of sign info
				if (x<0) x=-x;
				if (y<0) y=-y;
				if (z<0) z=-z;
		  		// Get rid of multiples of box lengths 
				//TODO  WIERD that should be a way to simplify it
				while (x > box[0]) x = x - box[0];
				while (y > box[1]) y = y - box[1];
				while (z > box[2]) z = z - box[2];

				//below is actually slower! 
				//x = x - box[0]*((int)x/box[0]);
				//y = y - box[0]*((int)y/box[1]);
				//z = z - box[0]*((int)z/box[2]);
		  	// Find shortest distance in periodic reference
				double D = box[0] - x;
				if (D < x) x = D;
				D = box[1] - y;
				if (D < y) y = D;  
				D = box[2] - z;
				if (D < z) z = D;


				//Dist = x*x + y*y + z*z;
				dist =  x*x + y*y + z*z;
				if (box[0]==0.0 || box[1]==0.0 || box[2]==0.0)
					dist = -1.0;


		


				//if (mol ==  11)
				//	printf("min  = %f\n",min_val);
				min_val = min(min_val,dist);


			}

			start = end;
			end = min(end + chunksize, NSAtoms*3);


		}

		dist_array[threadIdx.x] = min_val;
		//if (threadIdx.x == 0)
		//	printf("min_val  = %f\n",min_val);
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		//min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}





