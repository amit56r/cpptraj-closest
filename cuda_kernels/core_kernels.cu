
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM 1024


__global__ void Action_noImage_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms, int active_size)
{

	__shared__ double dist_array[BLOCKDIM];

	double Dist;


	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	//advantage of register
	double a0 = maskCenter[0];
	double a1 = maskCenter[1];
	double a2 = maskCenter[2];



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads();


		int sIndex =  mol*NAtoms*3 + atom*3;

		double x =  a0 - SolventMols_[sIndex + 0];
		double y = a1 - SolventMols_[sIndex + 1];
		double z =  a2 - SolventMols_[sIndex + 2];
	//Dist = x*x + y*y + z*z;
		dist_array[threadIdx.x] = x*x + y*y + z*z;
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		int i;
		// double min_val  = maxD;
		// if( threadIdx.x < active_size/NAtoms )
		// {

		// 	for(i  = threadIdx.x*NAtoms ; i <threadIdx.x*NAtoms + NAtoms ; i++ ){
		// 		//sIndex = mol*NAtoms*3 + i*3;
		// 		if (dist_array[i]  < min_val) 
		// 			min_val = dist_array[i] ;
		// 	}
		// 	D_[blockIdx.x * active_size/NAtoms + threadIdx.x  ] = min_val;
		// }


		double min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				if (dist_array[threadIdx.x + i]  < min_val) 
					min_val = dist_array[threadIdx.x + i] ;
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}


	// int i;
	// 	double min_val  = maxD;
	// 	if( atom ==0 )
	// 	{
	// 		for(i  = 0 ; i < NAtoms ; i++ ){
	// 			//sIndex = mol*NAtoms*3 + i*3;
	// 			if (dist_array[threadIdx.x + i]  < min_val) 
	// 				min_val = dist_array[threadIdx.x + i] ;
	// 		}
	// 		D_[mol] = min_val;
	// 	}