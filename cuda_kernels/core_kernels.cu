
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM 1024
#define RSIZE 1024

//------------------------------------------------------------------------------------------------------------------------------------------------
//try thread coarsening 

__global__ void Action_noImage_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms, int active_size)
{

	__shared__ double dist_array[BLOCKDIM];



	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	//advantage of register
	double a0 = maskCenter[0];
	double a1 = maskCenter[1];
	double a2 = maskCenter[2];



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads();


		int sIndex =  mol*NAtoms*3 + atom*3;

		double x =  a0 - SolventMols_[sIndex + 0];
		double y = a1 - SolventMols_[sIndex + 1];
		double z =  a2 - SolventMols_[sIndex + 2];
	//Dist = x*x + y*y + z*z;
		dist_array[threadIdx.x] = x*x + y*y + z*z;
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		int i;



		double min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}


	// int i;
	// 	double min_val  = maxD;
	// 	if( atom ==0 )
	// 	{
	// 		for(i  = 0 ; i < NAtoms ; i++ ){
	// 			//sIndex = mol*NAtoms*3 + i*3;
	// 			if (dist_array[threadIdx.x + i]  < min_val) 
	// 				min_val = dist_array[threadIdx.x + i] ;
	// 		}
	// 		D_[mol] = min_val;
	// 	}


	// double min_val  = maxD;
	// if( threadIdx.x < active_size/NAtoms )
	// {

	// 	for(i  = threadIdx.x*NAtoms ; i <threadIdx.x*NAtoms + NAtoms ; i++ ){
	// 		//sIndex = mol*NAtoms*3 + i*3;
	// 		if (dist_array[i]  < min_val) 
	// 			min_val = dist_array[i] ;
	// 	}
	// 	D_[blockIdx.x * active_size/NAtoms + threadIdx.x  ] = min_val;
	// }

//------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void Action_noImage_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, int Nmols , int NAtoms,int NSAtoms , int active_size)
{

	__shared__ double dist_array[BLOCKDIM];
	__shared__ double sAtom_shared[RSIZE];




	int mol  =  (blockIdx.x * active_size + threadIdx.x)/NAtoms; 
	int atom  = (blockIdx.x * active_size + threadIdx.x) - (mol * NAtoms);
	//int mol_in_block = threadIdx.x/NAtoms;

	

	//handling the chunks for  solute_atoms
	int chunksize,start,end, NChunks,i,j;

	if(NSAtoms*3 > RSIZE)
	{
		chunksize = (RSIZE/3)*3;
		NChunks = ceil(double(NSAtoms*3)/chunksize);
		start = 0;
		end = chunksize;
	}
	else
	{
		chunksize = NSAtoms*3;
		NChunks = 1;
		start = 0;
		end = NSAtoms*3;
	}

	// if(threadIdx.x == 0 && blockIdx.x == 0 )
	// 	printf("chunkszize = %d ; Nchunk =  %d; start = %d; end = %d\n ",
	// 		chunksize,NChunks,start,end);



	if ( threadIdx.x < active_size && mol*NAtoms + atom < Nmols*NAtoms )
	{

		// if(atom == 0 )
		// 	D_[mol] = maxD;
		//__syncthreads(); 
		double min_val  = maxD;
		double dist;
		int sIndex =  mol*NAtoms*3 + atom*3;
		double a0 = SolventMols_[sIndex + 0];
		double a1 = SolventMols_[sIndex + 1];
		double a2 = SolventMols_[sIndex + 2];


		for(i  = 0 ; i  < NChunks ; i++)
		{
			//copying to shared
			if (threadIdx.x < (end - start))
				sAtom_shared[threadIdx.x] = Solute_atoms[start + threadIdx.x];

			//TODO - add skew per thread 
			for (j = start ; j < end; j+=3 )
			{
				double x = Solute_atoms[j + 0]  - a0;
				double y = Solute_atoms[j + 1]  - a1;
				double z = Solute_atoms[j + 2]  - a2;
				dist =  x*x + y*y + z*z;
				//if (mol ==  11)
				//	printf("min  = %f\n",min_val);
				min_val = min(min_val,dist);


			}

			start = end;
			end = min(end + chunksize, NSAtoms*3);


		}

		dist_array[threadIdx.x] = min_val;
		//if (threadIdx.x == 0)
		//	printf("min_val  = %f\n",min_val);
	//printf(" dist  =  %f\n", Dist);

		__syncthreads();

	//first thread
	//naive approach to a reduction algorithm
	//this works if NAtoms is small other wise you need split
	//and do some of log(n) parallel reduction 
		//min_val  = maxD;
		if( atom ==0 )
		{
			for(i  = 0 ; i < NAtoms ; i++ ){
				//sIndex = mol*NAtoms*3 + i*3;
				//if (dist_array[threadIdx.x + i]  < min_val) 
				//	min_val = dist_array[threadIdx.x + i] ;
				min_val =  min(min_val, dist_array[threadIdx.x + i]);
			}
			D_[mol] = min_val;
		}

	//if(tx == 0 && bx == 0 )
	//	printf("end of kernel");
	}
}